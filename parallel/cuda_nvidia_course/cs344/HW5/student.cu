#include "hip/hip_runtime.h"
/* Udacity HW5
  Histogramming for Speed

  The goal of this assignment is compute a histogram
  as fast as possible.  We have simplified the problem as much as
  possible to allow you to focus solely on the histogramming algorithm.

  The input values that you need to histogram are already the exact
  bins that need to be updated.  This is unlike in HW3 where you needed
  to compute the range of the data and then do:
  bin = (val - valMin) / valRange to determine the bin.

  Here the bin is just:
  bin = val

  so the serial histogram calculation looks like:
  for (i = 0; i < numElems; ++i)
   histo[val[i]]++;

  That's it!  Your job is to make it run as fast as possible!

  1. Пользуемся распределением
  // Центральные значения обновляются много чаще
  The values are normally distributed - you may take
  advantage of this fact in your implementation.

  2. Базовая стратегия. 
  http://forums.udacity.com/questions/100040006/any-hw5-timing-results-on-a-solution-that-doesnt-simply-use-atomicadd-in-shared-and-global-mem#cs344
  Не ясно что с сортировкой - это же тоже время. и что сортировать то?
  И если много монипуляций, то может и не выйдет быстрее

*/


#include "utils.h"

static const int g_chunk = 4;
//__device__ __constant__   // падает произв
static const int d_chunk = g_chunk;

//static 
__global__ void src_histo_kernel(
    const unsigned int * const d_vals,
          unsigned int * const d_histo, 
    const unsigned int numBins,
    const unsigned int numElems)
{ 
  int g_id = threadIdx.x + blockDim.x * blockIdx.x;
  if (g_id >= numElems)
    return; 

  // bin
  unsigned int bin = d_vals[g_id];

  // Inc global memory. Partial histos not used.
  atomicAdd(&(d_histo[bin]), 1);
}

__global__
void yourHisto(const unsigned int* const vals, //INPUT
               unsigned int* const histo,      //OUPUT
               int numVals)
{
  //TODO fill in this kernel to calculate the histogram
  //as quickly as possible

  //Although we provide only one kernel skeleton,
  //feel free to use more if it will help you
  //write faster code
}

// Размер блока должен быть не меньше числа бинов
// 
static __global__ 
void histo_kernel_bins_shared_any_any(
    const unsigned int * const d_vals,
          unsigned int * const d_histo, 
    const unsigned int numBins,
    const unsigned int numElems)    
{ 
  extern   // если захардкодить размер, то быстрее не будет
  __shared__ unsigned int local_histo[];
  //__shared__ 
  unsigned slice[d_chunk];

  int g_id = blockDim.x * blockIdx.x + threadIdx.x;
  if (g_id >= numElems)
    return; 

  int ref = g_id * d_chunk;
  //if (ref+d_chunk >= numElems);
  //  return;


  for (int i = threadIdx.x; i < numBins; i += blockDim.x) 
    local_histo[i] = 0;

  for (int i = 0; i < d_chunk; ++i) {
    // bin
    const unsigned bin = d_vals[ref+i];  // read is coal.
    slice[i] = bin;
  }

  __syncthreads();

  // каждый поток обрабатывает отрезок
  for (int i = 0; i < d_chunk; ++i) {
    // bin
    //const unsigned bin = d_vals[ref+i];  // read is coal.

    // Inc global memory. Partial histos not used.
    // если поток столько же скольно и бинов, то гонок не будет
    //++local_histo[bin];  // значения могут быть одинаковыми
    atomicAdd(&(local_histo[slice[i]]), 1);
  }

  __syncthreads();  // ждем пока посчитают все потоки

  // Только первые потоки блока обновляеют гистограмму
  for (int i = threadIdx.x; i < numBins; i += blockDim.x) {
    const unsigned int value = local_histo[i];
      if (value)  // уменьшило время но не на много
        atomicAdd(&(d_histo[i]), value);  // write is coal.
  }
}

static __global__ 
void histo_kernel_bins_shared_any_tblock(
    const unsigned int * const d_vals,
          unsigned int * const d_histo, 
    const unsigned int numBins,
    const unsigned int numElems)    
{ 
  extern   // если захардкодить размер, то быстрее не будет
  __shared__ unsigned int local_histo[];

  int g_id = blockDim.x * blockIdx.x + threadIdx.x;
  //int tid = threadIdx.x;
  if (g_id >= numElems)
    return; 


  for (int i = threadIdx.x; i < numBins; i += blockDim.x) 
    local_histo[i] = 0;

  __syncthreads();

  // каждый поток обрабатывает отрезок
  //if (threadIdx.x % 8 == 0)
    //for (int i = 0, end = 1; i < end; ++i) {
      // bin
      unsigned int bin = d_vals[g_id+0];  // read is coal.

      // Inc global memory. Partial histos not used.
      // если поток столько же скольно и бинов, то гонок не будет
      //++local_histo[bin];  // значения могут быть одинаковыми
      atomicAdd(&(local_histo[bin]), 1);
    //}

  __syncthreads();  // ждем пока посчитают все потоки

  // Только первые потоки блока обновляеют гистограмму
  for (int i = threadIdx.x; i < numBins; i += blockDim.x) {
    const unsigned int value = local_histo[i];
      if (value)  // уменьшило время но не на много
        atomicAdd(&(d_histo[i]), value);  // write is coal.
  }
}

void computeHistogram(const unsigned int* const d_vals, //INPUT
                            unsigned int* const d_histo,      //OUTPUT
                      const unsigned int numBins,
                      const unsigned int numElems)
{
  //TODO Launch the yourHisto kernel
  // перемешанность данных балансирует нагрузку - если отсортировать вход - производительность резко упадет
  assert(numBins == 1024);
  //const int maxThreadsPerBlock = 1024;

	int threads = 1024;  // пробую меньше, чем число бинов
  int blocks = ceil((1.0f*numElems) / threads);
  //blocks /= 8;
  //int sub_hist_blocks = ceil((1.0f*numElems) / numBins);

  // и блок то меньше не сделать - будет не хватать индексов
  //assert(threads >= numBins);

  //thrust::sort(d_vals, d_vals + numElems);
  // Если отсортировать, то бины которые по середине будут писать практически в одно место - это ограничивает конкуренцию

  //if you want to use/launch more than one kernel,
  //feel free
  //src_histo_kernel<<< blocks, threads >>>(d_vals, d_histo, numBins, numElems);
  // 
  // Память может быть большей чем размеры блоков, просто правильно нужно будет обновить глоб. гист.
  //histo_kernel_bins_shared_any_tblock
  histo_kernel_bins_shared_any_any
  <<< 
    //blocks, threads/g_chunk, (numBins)* sizeof(unsigned int) /*+ 225*/ >>>
    blocks, threads/g_chunk, (numBins+4)* sizeof(unsigned int) + 225 >>>
    (d_vals, d_histo, numBins, numElems);
  //histo_kernel_bins_shared<<< sub_hist_blocks, numBins, numBins * sizeof(unsigned int) >>>(d_vals, d_histo, numBins, numElems);

  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}
