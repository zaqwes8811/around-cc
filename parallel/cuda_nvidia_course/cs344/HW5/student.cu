#include "hip/hip_runtime.h"
/* Udacity HW5
   Histogramming for Speed

   The goal of this assignment is compute a histogram
   as fast as possible.  We have simplified the problem as much as
   possible to allow you to focus solely on the histogramming algorithm.

   The input values that you need to histogram are already the exact
   bins that need to be updated.  This is unlike in HW3 where you needed
   to compute the range of the data and then do:
   bin = (val - valMin) / valRange to determine the bin.

   Here the bin is just:
   bin = val

   so the serial histogram calculation looks like:
   for (i = 0; i < numElems; ++i)
     histo[val[i]]++;

   That's it!  Your job is to make it run as fast as possible!

   The values are normally distributed - you may take
   advantage of this fact in your implementation.

*/


#include "utils.h"

template <class Type> __device__ __host__ Type cuda_min2( Type a, Type b ) {
  // I - +inf
  return a < b ? a : b;
}

template <class Type> __device__ __host__ Type cuda_max2( Type a, Type b ) {
  // I - -inf
  return a > b ? a : b;
}

//static 
__global__ void src_histo_kernel(
    const unsigned int * const d_vals,
          unsigned int * const d_histo, 
    const unsigned int numBins,
    const unsigned int numElems
    //float min_logLum, float logLumRange
    )
{ 
  int g_id = threadIdx.x + blockDim.x * blockIdx.x;
  if (g_id >= numElems)
    return; 
    
  //float value = d_vals[g_id];

  // bin
  unsigned int bin = d_vals[g_id];
  //cuda_min2(
  //    static_cast<unsigned int>(numBins - 1), 
  //    static_cast<unsigned int>((value - min_logLum) / logLumRange * numBins));

  // Inc global memory. Partial histos not used.
  atomicAdd(&(d_histo[bin]), 1);
}

__global__
void yourHisto(const unsigned int* const vals, //INPUT
               unsigned int* const histo,      //OUPUT
               int numVals)
{
  //TODO fill in this kernel to calculate the histogram
  //as quickly as possible

  //Although we provide only one kernel skeleton,
  //feel free to use more if it will help you
  //write faster code
}

void computeHistogram(const unsigned int* const d_vals, //INPUT
                            unsigned int* const d_histo,      //OUTPUT
                      const unsigned int numBins,
                      const unsigned int numElems)
{
  //TODO Launch the yourHisto kernel
  const int maxThreadsPerBlock = 1024;
	int threads = maxThreadsPerBlock;
  int blocks = ceil((1.0f*numElems) / maxThreadsPerBlock);

  //if you want to use/launch more than one kernel,
  //feel free
  src_histo_kernel<<< blocks, threads >>>(d_vals, d_histo, numBins, numElems);

  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}
